#include "hip/hip_runtime.h"

/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include <fstream>
#include <string>
#include <iostream>

#include <hip/hip_runtime_api.h>


#include "getColor.h"
#include "getName140.h"
#include "getArrayColor.h"
#include "getArrayColor140.h"

static void CheckCudaErrorAux(const char *, unsigned,const char*, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)



static void CheckCudaErrorAux(const char *file, unsigned line, const char *statement, hipError_t err){
	if(err == hipSuccess) return;
	std::cerr<< statement << " returned " << hipGetErrorString(err) << "("
			<< err << ") at " << file << ":" << line << std::endl;
	exit(1);
}

unsigned long *generate_data(int size){
	unsigned long *data = (unsigned long *) malloc(sizeof(unsigned long) * size);
	std::string line;
	std::ifstream file ("colors.txt");
	int i = 0;
	if(file.is_open()){
		while(std::getline(file,line)  && i < size){
			data[i] = std::strtoul(line.c_str(),NULL, 16);
			i++;
		}
	}
	return data;
}
void writeOut(unsigned long *data , unsigned int dim){
	for(int i = 0; i < dim; i++){

		printf("Color %d: %s=> %lu\n",i, getName140(data[i*2]).c_str() , data[i*2+1]);
	}

}
unsigned long *prepareOutput(int size){
	unsigned long *data = (unsigned long *) malloc(size * size * sizeof(unsigned long));
	for(int i = 0; i<size * size; i++){
		data[i] = 0;
	}

	return data;
}

void sortOut(unsigned long *data , unsigned int dim){
	for (unsigned int i = 0; i < dim; i++) {
		unsigned int posMax = i;
		for (unsigned int k = i; k < dim; k++) {
			if(data[k*2+1] > data[posMax*2+1]){
				posMax = k;
			}
		}

		if (posMax != i) {
			unsigned long tempHex = data[i*2];
			unsigned long tempCount = data[i*2+1];

			data[i*2] = data[posMax*2];
			data[i*2+1] = data[posMax*2+1];

			data[posMax*2] = tempHex;
			data[posMax*2+1] = tempCount;
		}
	}
}





/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 *
 * See hip/hip_runtime.h for error code descriptions.
 */

__global__ void colorCount(unsigned long* vectArrayColor, unsigned long* vectColor,unsigned long* vectRisu, unsigned int size , unsigned int sizeColor){

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < sizeColor) {
		long sum = -1;
		for (int ii = 0; ii < size; ii++) {
			if(vectArrayColor[index] == vectArrayColor[ii]){
				sum = sum + 1;
			}
		}
		vectRisu[index * 2] = vectArrayColor[index];
		vectRisu[index * 2 + 1] = sum;
	}

}

int main(int argc, char **argv)
{
	unsigned int size = 276;
	unsigned int sizeColor = 140;
	unsigned long *hostArrayColor = getArrayColor140();
	unsigned long *hostColor;
	unsigned long *hostRisu;
	unsigned long *deviceArrayColor;
	unsigned long *deviceColor;
	unsigned long *deviceRisu;

	hostColor = generate_data(size);
	hostRisu = prepareOutput(sizeColor);

	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&deviceArrayColor,
					sizeof(unsigned long) * sizeColor));
	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&deviceColor,
					sizeof(unsigned long) * size));
	CUDA_CHECK_RETURN(
			hipMalloc((void ** )&deviceRisu,
					sizeof(unsigned long) * sizeColor * sizeColor));

	//copy dataHost to datatDevice
	CUDA_CHECK_RETURN(
			hipMemcpy(deviceArrayColor,hostArrayColor, sizeColor * sizeof(unsigned long),
					hipMemcpyHostToDevice));

	CUDA_CHECK_RETURN(
			hipMemcpy(deviceColor,hostColor, size * sizeof(unsigned long),
					hipMemcpyHostToDevice));

	CUDA_CHECK_RETURN(
			hipMemcpy(deviceRisu,hostRisu, sizeColor * sizeColor * sizeof(unsigned long),
					hipMemcpyHostToDevice));


	colorCount<<<1,sizeColor>>>(deviceArrayColor,deviceColor , deviceRisu , size, sizeColor);

	hipDeviceSynchronize();

	CUDA_CHECK_RETURN(
			hipMemcpy(hostRisu,deviceRisu, sizeColor * sizeColor * sizeof(unsigned long),
					hipMemcpyDeviceToHost));


	//Free GPU
	hipFree(deviceArrayColor);
	hipFree(deviceColor);
	hipFree(deviceRisu);
	sortOut(hostRisu,sizeColor);
	//writeOut(hostRisu,*hostRisuDim);

	writeOut(hostRisu,sizeColor);
	//Free host memory
	free(hostArrayColor);
	free(hostColor);
	free(hostRisu);

	return 0;
}
